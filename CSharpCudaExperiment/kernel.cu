#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include <stdio.h>

extern "C" {
    __global__ void FactorKernel(int* m, int v)
    {
        //int i = threadIdx.x + (blockDim.x * blockIdx.x);
        //m[i] = v*i;
        m[threadIdx.x + (blockDim.x * blockIdx.x)] *= v;
    }

    __global__ void SetKernel(int* m, int v)
    {
        m[threadIdx.x + (blockDim.x * blockIdx.x)] = v;
    }

    __global__ void AddKernel(int* m, int v)
    {
        m[threadIdx.x + (blockDim.x * blockIdx.x)] += v;
    }

    __global__ void GetEnergy(float* x, float* y, float* z, int i)
    {
        //http://cuda-programming.blogspot.com/2013/01/vector-dot-product-in-cuda-c-cuda-c.html
    }

    __global__ void SequenceProduct (const int N, const float* V1, const float* V2, float* V3)
    {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid < N)
            V3[tid] = V1[tid] * V2[tid];
    }

    __global__ void VectorSum(const int N, const float* v, float * sum)
    {
        __shared__ float chache[1024];
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        const unsigned int chacheindex = threadIdx.x;

        float temp = 0;
        while (tid < N)
        {
            temp += v[tid];
            tid += blockDim.x * gridDim.x;
        }
        chache[chacheindex] = temp;
        __syncthreads();

        int i = blockDim.x / 2;
        while (i != 0)
        {
            if (chacheindex < i)
                chache[chacheindex] += chache[chacheindex + i];
            __syncthreads();
            i /= 2;
        }
        if (chacheindex == 0)
            sum[blockIdx.x] = chache[0];
    }

    __global__ void VectorDotProduct (const int N, const float* V1, const float* V2, float* V3)
    {
        __shared__ float chache[1024];
        float temp;
        unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        const unsigned int chacheindex = threadIdx.x;

        while (tid < N)
        {
            temp += V1[tid] * V2[tid];
            tid += blockDim.x * gridDim.x;
        }
        chache[chacheindex] = temp;
        __syncthreads();

        int i = blockDim.x / 2;
        while (i != 0)
        {
            if (chacheindex < i)
                chache[chacheindex] += chache[chacheindex + i];
            __syncthreads();
            i /= 2;
        }
        if (chacheindex == 0)
            V3[blockIdx.x] = chache[0];
    }
}

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Name:                          %s\n", devProp.name);
    printf("Total global memory:           %u\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    for (int i = 1; i <= 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i - 1]);
    for (int i = 1; i <= 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i - 1]);
    printf("Clock rate:                    %d\n", devProp.clockRate);
    printf("Total constant memory:         %u\n", devProp.totalConstMem);
    printf("Texture alignment:             %u\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int main()
{
    // Number of CUDA devices
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    // Iterate through devices
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    printf("\nPress any key to exit...");
    char c;
    scanf("%c", &c);

    return 0;
}